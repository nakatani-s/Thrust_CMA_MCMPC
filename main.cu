#include "hip/hip_runtime.h"
#include<iostream>
#include <stdio.h>
#include <fstream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <errno.h>
#include <string.h>
#include <unistd.h>
#include <iomanip>

#include "include/params.cuh"
#include "include/DataStructure.cuh"
#include "include/MCMPC.cuh"
#include "include/init.cuh"
#include "include/cuSolverForMCMPC.cuh"

#define Linear

void printMatrix(int m, int n, float*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            float Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
            //printf("%s[%d] = %f\n", name, row + col*lda, Areg);
        }
    }
}
int main(int argc, char **argv)
{
    hipsolverHandle_t cusolverH = NULL;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;

    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    /*データ書き込みファイルの定義*/
    FILE *fp;
    time_t timeValue;
    struct tm *timeObject;
    time( &timeValue );
    timeObject = localtime( &timeValue );
    char filename1[35];
    sprintf(filename1,"data_system_%d%d_%d%d.txt",timeObject->tm_mon + 1, timeObject->tm_mday, timeObject->tm_hour,timeObject->tm_min);
    fp = fopen(filename1,"w");


    float params[dim_param], state[dim_state], /*h_constraint[NUM_CONST],*/ h_matrix[dim_weight_matrix];
    float *device_param, *device_matrix;
    Mat_sys_A( params );
    init_state( state );
    // init_constraint( h_constraint );
    init_Weight_matrix( h_matrix );
    //hipMemcpyToSymbol(HIP_SYMBOL(d_param), &params, dim_param * sizeof(float));
    hipMalloc(&device_param, sizeof(float)*dim_param);
    hipMalloc(&device_matrix, sizeof(float)*dim_weight_matrix);
    hipMemcpy(device_param, params, sizeof(float)*dim_param, hipMemcpyHostToDevice);
    hipMemcpy(device_matrix, h_matrix, sizeof(float)*dim_weight_matrix, hipMemcpyHostToDevice);
    

#ifdef Linear
    float opt[HORIZON], Error[HORIZON];
    init_opt( opt );
#endif


    /* GPUの設定 */
    unsigned int numBlocks, randomBlocks, randomNums/*, minId_cpu*/;
    int Blocks;
    randomNums = N_OF_SAMPLES * (dim_U+1) * HORIZON;
    randomBlocks = countBlocks(randomNums, THREAD_PER_BLOCKS);
    numBlocks = countBlocks(N_OF_SAMPLES, THREAD_PER_BLOCKS);
    printf("#NumBlocks = %d\n", numBlocks);
    Blocks = numBlocks;

    /* CPU to GPU dataExchanger */
    Data1 *h_dataFromBlocks;
    Data1 *d_dataFromBlocks;

#ifdef USING_THRUST
    thrust::device_vector<int> indices_device_vec( N_OF_SAMPLES );
    //thrust::device_vector<int> indices_vec_dev_temp( N_OF_SAMPLES );
    //indices_device_vec = indices_vec_dev_temp;
    thrust::device_vector<float> cost_device_vec_for_sorting( N_OF_SAMPLES );
    //thrust::device_vector<float> cost_vec_dev_temp( N_OF_SAMPLES );
    //cost_device_vec_for_sorting = cost_vec_dev_temp;
    
    Input_vec *d_Input_vec;
    //Input_vec *h_Input_vec;
    //h_Input_vec = (Input_vec *)malloc(sizeof(Input_vec) * N_OF_SAMPLES);
    hipMalloc(&d_Input_vec, sizeof(Input_vec) * N_OF_SAMPLES);
    set_Input_vec<<<N_OF_SAMPLES,1>>>(d_Input_vec, 0.0f);

    /*for(int i = 0; i < N_OF_SAMPLES; i++){
        for(int k = 0; k < HORIZON; k++){
            h_Input_vec[i].Input[k] = 0.0f;
        }
    }*/
#endif
    /*Data1 *h_dataFromBlocks;
    Data1 *d_dataFromBlocks;*/
    h_dataFromBlocks = (Data1 *)malloc(sizeof(Data1)*numBlocks);
    hipMalloc(&d_dataFromBlocks, sizeof(Data1) * numBlocks);



    /* hiprand の設定 */
    hiprandState *devStates;
    hipMalloc((void **)&devStates, randomNums * sizeof(hiprandState));
    setup_kernel<<<N_OF_SAMPLES * (dim_U+1), HORIZON>>>(devStates,rand());
    hipDeviceSynchronize();

    /* Covariance の定義 */
    float *h_hat_Q, *Diag_D;
    float *device_cov;
    float *device_diag_eig = NULL;
    float *d_hat_Q;
    h_hat_Q = (float *)malloc(sizeof(float)*dim_hat_Q);
    Diag_D = (float *)malloc(sizeof(float)*dim_hat_Q);
    hipMalloc(&device_cov, sizeof(float)*dim_hat_Q);
    hipMalloc(&device_diag_eig, sizeof(float)*dim_hat_Q);
    hipMalloc(&d_hat_Q, sizeof(float)*dim_hat_Q);

    setup_init_Covariance<<<HORIZON, HORIZON>>>(d_hat_Q);

    /* 準最適制御入力列 */
    float *Us_host, *Us_device;
    Us_host = (float *)malloc(sizeof(float) * HORIZON);
    for(int i = 0; i < HORIZON; i++){
        Us_host[i] = 0.0f;
    }
    hipMalloc(&Us_device, sizeof(float) * HORIZON);


    float var;
    float now_u;
    for(int i = 0; i < Blocks; i++){
        for(int k = 0; k < HORIZON; k++){
            h_dataFromBlocks[i].Input[k] = 0.0f;
        }
    }

    /* 固有値の取得 */
    

    const int m = HORIZON;
    const int lda = m;

    float eig_vec[m] = { };

    float *d_A;
    float *d_W;
    int *devInfo;
    float *d_work;
    int lwork = 0;

    // int work_size;
    // float *work_space;

    int info_gpu = 0;

    cudaStat1 = hipMalloc ((void**)&d_A, sizeof(float) * lda * m);
    cudaStat2 = hipMalloc ((void**)&d_W, sizeof(float) * m);
    cudaStat3 = hipMalloc ((void**)&devInfo, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;



    for(int time = 0; time < TIME; time++){
        for(int repeat = 0; repeat < Recalc; repeat++){
            var = Variavility * pow(0.8,repeat);
            //var = Variavility;
            hipMemcpy(d_dataFromBlocks, h_dataFromBlocks, sizeof(Data1)*numBlocks, hipMemcpyHostToDevice);
            hipDeviceSynchronize();
            // MCMPC_GPU<<<numBlocks, THREAD_PER_BLOCKS>>>(state, devStates, d_dataFromBlocks, var, Blocks, d_hat_Q);
#ifdef USING_THRUST
            Using_Thrust_MCMPC_Linear<<<numBlocks, THREAD_PER_BLOCKS>>>(state[0],state[1],state[2],devStates, d_Input_vec, var, Blocks, d_hat_Q, device_param, device_matrix, thrust::raw_pointer_cast( cost_device_vec_for_sorting.data() ));
            thrust::sequence( indices_device_vec.begin(), indices_device_vec.end() );
            thrust::sort_by_key( cost_device_vec_for_sorting.begin(), cost_device_vec_for_sorting.end(), indices_device_vec.begin() );
            callback_elite_sample<<<Blocks,1>>>(d_dataFromBlocks, d_Input_vec, thrust::raw_pointer_cast( indices_device_vec.data() ));
#else
            MCMPC_GPU_Linear_Example<<<numBlocks, THREAD_PER_BLOCKS>>>(state[0],state[1],state[2], devStates, d_dataFromBlocks, var, Blocks, d_hat_Q, device_param, device_matrix);
            hipDeviceSynchronize();
            //hipMemcpy(h_dataFromBlocks, d_dataFromBlocks, sizeof(Data1) * numBlocks, hipMemcpyDeviceToHost);
#endif
            hipMemcpy(h_dataFromBlocks, d_dataFromBlocks, sizeof(Data1) * numBlocks, hipMemcpyDeviceToHost);
            printf("TOP  W == %f WORST W == %f\n",h_dataFromBlocks[0].W,  h_dataFromBlocks[Blocks-1].W);
            weighted_mean(h_dataFromBlocks, Blocks, Us_host);
            //printMatrix(m,1,Us_host, m, "u");
            hipMemcpy(Us_device, Us_host, sizeof(float) * HORIZON, hipMemcpyHostToDevice);
            //printf("hoge\n");
            calc_Var_Cov_matrix<<<HORIZON, HORIZON>>>(device_cov, d_dataFromBlocks, Us_device, Blocks);
            hipDeviceSynchronize();
            hipMemcpy(h_hat_Q, device_cov, sizeof(float)*dim_hat_Q, hipMemcpyDeviceToHost);
            //printMatrix(m,m,h_hat_Q, lda, "DA");
            
            //cudaStat1 = hipMemcpy(d_A, h_hat_Q, sizeof(float) * lda * m, hipMemcpyHostToDevice);
            cudaStat1 = hipMemcpy(d_A, h_hat_Q, sizeof(float) * lda * m, hipMemcpyHostToDevice);
            assert(hipSuccess == cudaStat1);
            cusolver_status = hipsolverDnSsyevd_bufferSize(
                cusolverH,
                jobz,
                uplo,
                m,
                d_A,
                lda,
                d_W,
                &lwork);
            assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
            //hipMemcpy(Diag_D, d_A, sizeof(float)*lda*m, hipMemcpyDeviceToHost);
            //printMatrix(m,m,Diag_D, m, "V");
            

            cudaStat1 = hipMalloc((void**)&d_work, sizeof(float)*lwork);
            //assert(hipSuccess == cudaStat1);

            cusolver_status = hipsolverDnSsyevd(
                cusolverH,
                jobz,
                uplo,
                m,
                d_A,
                lda,
                d_W,
                d_work,
                lwork,
                devInfo);

            hipDeviceSynchronize();
            //assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
            //assert(hipSuccess == cudaStat1);

            cudaStat1 = hipMemcpy(eig_vec, d_W, sizeof(float)*m, hipMemcpyDeviceToHost);
            cudaStat2 = hipMemcpy(Diag_D, d_A, sizeof(float)*lda*m, hipMemcpyDeviceToHost);
            cudaStat3 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
            //assert(hipSuccess == cudaStat1);
            //assert(hipSuccess == cudaStat2);
            //assert(hipSuccess == cudaStat3);
            // printMatrix(m,1,eig_vec, lda, "C");
            // printf("=====Upper is eigen value====");
            //printMatrix(m,m,Diag_D, lda, "C");
            make_Diagonalization<<<HORIZON,HORIZON>>>(d_W, d_A);
            hipMemcpy(h_hat_Q, d_A, sizeof(float)*lda*m, hipMemcpyDeviceToHost);
            //printMatrix(m,m,h_hat_Q, lda, "C");
            hipMemcpy(device_diag_eig, h_hat_Q, sizeof(float)*dim_hat_Q, hipMemcpyHostToDevice);
            hipMemcpy(d_hat_Q, Diag_D, sizeof(float)*dim_hat_Q, hipMemcpyHostToDevice);
            tanspose<<<HORIZON,HORIZON>>>(device_cov, d_hat_Q);
            pwr_matrix_answerB<<<HORIZON,HORIZON>>>(device_cov, device_diag_eig);
            hipDeviceSynchronize();

            /*cusolver_status = hipsolverDnSpotrf_bufferSize(cusolverH, uplo, m, device_cov, m, &work_size);
            assert( cusolver_status == HIPSOLVER_STATUS_SUCCESS );
            // float* workspace ;
            hipMalloc((void**)&work_space, sizeof(float)*work_size);
            cusolver_status = hipsolverDnSpotrf(cusolverH, uplo, m, device_cov, m , work_space, work_size, devInfo);
            assert( cusolver_status == HIPSOLVER_STATUS_SUCCESS );
            setup_init_Covariance<<<HORIZON, HORIZON>>>(d_hat_Q);
            cusolver_status = hipsolverDnSpotrs(cusolverH, uplo, m, m , device_cov, m, d_hat_Q, m, devInfo);
            assert( cusolver_status == HIPSOLVER_STATUS_SUCCESS );*/
            tanspose<<<HORIZON,HORIZON>>>(d_hat_Q, device_cov);
            // pwr_matrix_answerA<<<HORIZON,HORIZON>>>(device_diag_eig, device_cov);
            pwr_matrix_answerA<<<HORIZON,HORIZON>>>(device_diag_eig, d_hat_Q);
            hipDeviceSynchronize();
            tanspose<<<HORIZON,HORIZON>>>(d_hat_Q, device_diag_eig);
            //hipMemcpy(h_hat_Q, device_diag_eig, sizeof(float)*dim_hat_Q, hipMemcpyDeviceToHost);
            hipMemcpy(h_hat_Q, d_hat_Q, sizeof(float)*dim_hat_Q, hipMemcpyDeviceToHost);
            //hipMemcpy(d_hat_Q, h_hat_Q, sizeof(float)*dim_hat_Q, hipMemcpyHostToDevice);
            printMatrix(m,m,h_hat_Q, lda, "C");

            fprintf(fp,"%f %f %f %f %f %f %f %f %f %f\n",Us_host[0], Us_host[1],
                    Us_host[2], Us_host[3], Us_host[4], Us_host[5], Us_host[6], Us_host[7], Us_host[8], Us_host[9]);

            for(int count = 0; count < HORIZON; count++){
                h_dataFromBlocks[0].Input[count] = Us_host[count];
            }
#ifdef USING_THRUST
                reset_Input_vec<<<numBlocks,THREAD_PER_BLOCKS>>>(d_Input_vec, Us_device);
                hipDeviceSynchronize();
#endif

#ifdef Linear
            float RSME=0.0f;
            for(int d = 0; d < HORIZON; d++){
                Error[d] = Us_host[d] - opt[d];
                RSME += powf(Error[d],2);
            }
            printf("RSME == %f\n", RSME / HORIZON);
#endif
        }
        //printMatrix(m,m,h_hat_Q, lda, "C");
        now_u = Us_host[0];
        calc_Linear_example(state, now_u, params, state);
        for(int i = 0; i < Blocks; i++){
            for(int k = 0; k < HORIZON - 1; k++){
                h_dataFromBlocks[i].Input[k] = Us_host[k+1];
            }
            h_dataFromBlocks[i].Input[HORIZON-1] = Us_host[HORIZON - 1];
        }
    }
    if (d_A    ) hipFree(d_A);
    if (d_W    ) hipFree(d_W);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);
    //hipFree(indices_device_vec);
    //hipFree(cost_device_vec_for_sorting);

    if (cusolverH) hipsolverDnDestroy(cusolverH);
    fclose(fp);
    // fclose(hp);
    //thrust::reduce(indices_device_vec,cost_device_vec_for_sorting);
    hipDeviceReset();
    printf("%s\n", hipGetErrorString(hipGetLastError()));
    return 0;
}
